#include <stdio.h>
#include <hip/hip_runtime.h>

#define DATA_SIZE ( 1024 * 1024 * 1024 )  // 1GB
#define NUM_ITER 100

// CUDA错误检查宏
#define checkCudaError(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

// 测试主机到设备传输
float test_host_to_device(float* h_data, float* d_data, hipStream_t stream) {
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start));
    checkCudaError(hipEventCreate(&stop));

    // 预热
    checkCudaError(hipMemcpyAsync(d_data, h_data, DATA_SIZE, hipMemcpyHostToDevice, stream));
    
    checkCudaError(hipEventRecord(start, stream));
    for (int i = 0; i < NUM_ITER; ++i) {
        checkCudaError(hipMemcpyAsync(d_data, h_data, DATA_SIZE, hipMemcpyHostToDevice, stream));
    }
    checkCudaError(hipEventRecord(stop, stream));
    checkCudaError(hipEventSynchronize(stop));

    float milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop));
    
    checkCudaError(hipEventDestroy(start));
    checkCudaError(hipEventDestroy(stop));
    
    return milliseconds / NUM_ITER;  // 返回平均时间
}

// 测试设备到主机传输
float test_device_to_host(float* h_data, float* d_data, hipStream_t stream) {
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start));
    checkCudaError(hipEventCreate(&stop));

    // 预热
    checkCudaError(hipMemcpyAsync(h_data, d_data, DATA_SIZE, hipMemcpyDeviceToHost, stream));
    
    checkCudaError(hipEventRecord(start, stream));
    for (int i = 0; i < NUM_ITER; ++i) {
        checkCudaError(hipMemcpyAsync(h_data, d_data, DATA_SIZE, hipMemcpyDeviceToHost, stream));
    }
    checkCudaError(hipEventRecord(stop, stream));
    checkCudaError(hipEventSynchronize(stop));

    float milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop));
    
    checkCudaError(hipEventDestroy(start));
    checkCudaError(hipEventDestroy(stop));
    
    return milliseconds / NUM_ITER;  // 返回平均时间
}

// 新增分页内存传输测试函数
float test_pageable_transfer(void* src, void* dst, hipMemcpyKind kind) {
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start));
    checkCudaError(hipEventCreate(&stop));

    // 同步预热
    checkCudaError(hipMemcpy(dst, src, DATA_SIZE, kind));
    
    checkCudaError(hipEventRecord(start));
    for (int i = 0; i < NUM_ITER; ++i) {
        checkCudaError(hipMemcpy(dst, src, DATA_SIZE, kind));
    }
    checkCudaError(hipEventRecord(stop));
    checkCudaError(hipEventSynchronize(stop));

    float milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop));
    
    checkCudaError(hipEventDestroy(start));
    checkCudaError(hipEventDestroy(stop));
    
    return milliseconds / NUM_ITER;
}

int main() {
    float *h_data, *d_data, *h_pageable;
    hipStream_t stream;
    
    // 分配pinned host内存
    checkCudaError(hipHostMalloc(&h_data, DATA_SIZE, hipHostMallocDefault));
    // 分配分页host内存
    h_pageable = (float*)malloc(DATA_SIZE);
    if (!h_pageable) {
        fprintf(stderr, "Failed to allocate pageable host memory\n");
        exit(EXIT_FAILURE);
    }
    // 分配device内存
    checkCudaError(hipMalloc(&d_data, DATA_SIZE));
    // 创建CUDA流
    checkCudaError(hipStreamCreate(&stream));

    // 测试Host->Device
    float h2d_time = test_host_to_device(h_data, d_data, stream);
    printf("Host->Device 传输时间: %.3f ms\n带宽: %.2f GB/s\n", 
           h2d_time, 
           (DATA_SIZE / (1024.0f * 1024.0f * 1024.0f)) / (h2d_time / 1000.0f));

    // 测试Device->Host
    float d2h_time = test_device_to_host(h_data, d_data, stream);
    printf("\nDevice->Host 传输时间: %.3f ms\n带宽: %.2f GB/s\n", 
           d2h_time, 
           (DATA_SIZE / (1024.0f * 1024.0f * 1024.0f)) / (d2h_time / 1000.0f));

    // 新增分页内存测试
    float h2d_page_time = test_pageable_transfer(h_pageable, d_data, hipMemcpyHostToDevice);
    printf("\nPageable->Device 传输时间: %.3f ms\n带宽: %.2f GB/s",
           h2d_page_time, 
           (DATA_SIZE / (1024.0f * 1024.0f * 1024.0f)) / (h2d_page_time / 1000.0f));

    float d2h_page_time = test_pageable_transfer(d_data, h_pageable, hipMemcpyDeviceToHost);
    printf("\n\nDevice->Pageable 传输时间: %.3f ms\n带宽: %.2f GB/s",
           d2h_page_time,
           (DATA_SIZE / (1024.0f * 1024.0f * 1024.0f)) / (d2h_page_time / 1000.0f));

    // 新增分页内存释放
    free(h_pageable);
    // 清理资源
    checkCudaError(hipHostFree(h_data));
    checkCudaError(hipFree(d_data));
    checkCudaError(hipStreamDestroy(stream));

    return 0;
}
